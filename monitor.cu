
#include <hip/hip_runtime.h>
#include <ncurses.h>
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <nvml.h>
#include <ctime>
#include <vector>
#include <chrono>
#include <thread>

#define CHECK_NVML(result, message)                                           \
    if (result != NVML_SUCCESS)                                               \
    {                                                                         \
        endwin();                                                             \
        std::cerr << message << ": " << nvmlErrorString(result) << std::endl; \
        exit(1);                                                              \
    }

int main()
{
    // Initialize ncurses and colors
    initscr();
    noecho();
    curs_set(FALSE);
    start_color();
    init_pair(1, COLOR_RED,   COLOR_BLACK);   // TX in red
    init_pair(2, COLOR_GREEN, COLOR_BLACK);     // RX in green

    // Initialize NVML
    CHECK_NVML(nvmlInit(), "Failed to initialize NVML");

    unsigned int device_count;
    CHECK_NVML(nvmlDeviceGetCount(&device_count), "Failed to get device count");

    if (device_count == 0)
    {
        endwin();
        std::cerr << "No NVIDIA devices found." << std::endl;
        return 1;
    }

    // Open file for logging data
    std::ofstream outfile("bandwidth_data.txt");
    if (!outfile.is_open())
    {
        endwin();
        std::cerr << "Failed to open output file!" << std::endl;
        return 1;
    }

    // Get handle for the first device (adjust if needed)
    nvmlDevice_t device;
    CHECK_NVML(nvmlDeviceGetHandleByIndex(0, &device), "Failed to get device handle");

    // Get terminal dimensions
    int term_height, term_width;
    getmaxyx(stdscr, term_height, term_width);

    // Define the graph area (reserve space for y-axis labels and header/footer)
    const int graph_x_offset = 8;  // left margin for labels
    int plot_width  = term_width - graph_x_offset;
    int plot_height = term_height - 3; // header and footer rows

    // Sliding window buffers: one value per column of the plot area
    std::vector<unsigned int> tx_values(plot_width, 0);
    std::vector<unsigned int> rx_values(plot_width, 0);

    // Define maximum throughput (in KB/s) for scaling (adjust as needed)
    const unsigned int max_throughput = 30000;

    while (true)
    {
        // Fetch throughput values from NVML
        unsigned int tx_throughput, rx_throughput;
        CHECK_NVML(nvmlDeviceGetPcieThroughput(device, NVML_PCIE_UTIL_TX_BYTES, &tx_throughput),
                   "Failed to get PCIe TX throughput");
        CHECK_NVML(nvmlDeviceGetPcieThroughput(device, NVML_PCIE_UTIL_RX_BYTES, &rx_throughput),
                   "Failed to get PCIe RX throughput");

        // Log data with timestamp to file
        std::time_t now = std::time(0);
        char timestamp[20];
        std::strftime(timestamp, sizeof(timestamp), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
        outfile << timestamp << "," << tx_throughput << "," << rx_throughput << std::endl;

        // Update sliding window buffers: remove the oldest and append the new values
        tx_values.erase(tx_values.begin());
        rx_values.erase(rx_values.begin());
        tx_values.push_back(tx_throughput);
        rx_values.push_back(rx_throughput);

        // Clear the screen for redrawing
        clear();

        // Draw header
        mvprintw(0, 0, "PCIe Throughput Graph (TX in RED, RX in GREEN)");

        // Draw Y-axis labels on the left side
        for (int i = 0; i <= plot_height; i++)
        {
            double frac = (double)i / plot_height;
            unsigned int value = max_throughput - (unsigned int)(frac * max_throughput);
            mvprintw(i + 1, 0, "%5u|", value);
        }

        // Plot discrete points: one point per sample without connecting lines
        for (int x = 0; x < plot_width; x++)
        {
            int tx_row = 1 + (int)(((double)(max_throughput - tx_values[x]) / max_throughput) * plot_height);
            int rx_row = 1 + (int)(((double)(max_throughput - rx_values[x]) / max_throughput) * plot_height);

            attron(COLOR_PAIR(1));
            mvaddch(tx_row, x + graph_x_offset, '*');
            attroff(COLOR_PAIR(1));

            attron(COLOR_PAIR(2));
            mvaddch(rx_row, x + graph_x_offset, '+');
            attroff(COLOR_PAIR(2));
        }

        // Draw footer with the latest readings
        mvprintw(term_height - 1, 0, "Latest: TX = %u KB/s, RX = %u KB/s", tx_throughput, rx_throughput);

        // Refresh the screen to update the display
        refresh();

        // Short delay before the next update
        std::this_thread::sleep_for(std::chrono::milliseconds(20));
    }

    // Cleanup (unreachable in this infinite loop but good practice)
    outfile.close();
    nvmlShutdown();
    endwin();
    return 0;
}
